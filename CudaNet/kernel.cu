#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "Image_to_Matrix.h"
using namespace std;

__global__ void same(double* dev_matrix1,double* dev_kernel,double*dev_matrix2)
{
	int idx = threadIdx.x;
	int idy = threadIdx.y;
	int idz = threadIdx.z;
}

 __host__ void same_conv(double Matrix[m_sizex][m_sizey],double Matrix2[m_sizex][m_sizey][sizez], double kernel[k_sizex][k_sizey][sizez])
{
	 double* matrixline=new double[m_sizex*m_sizey];//�������� ������ ��� ������� �������,  ���������� ������ 
	 double* kernelline=new double[k_sizex*k_sizey*sizez];//�������� ������ ���  ����, ���������� ������
	 for (int x = 0; x < m_sizex; x++)//�������� ������� ������� 28*28 � ���������� ������ �� 28*28
	 {
		 for (int y = 0; y < m_sizey; y++)
		 {
			 matrixline[x * m_sizey + y] = Matrix[x][y];
		 }
	 }
	 for (int x = 0; x < k_sizex; x++)//�������� ���� �������� 5*5*20 � ���������� ������ 5*5*20
	 {
		 for (int y = 0; y < k_sizey; y++)
		 {
			 for (int z = 0; z < sizez; z++)
			 {
				 kernelline[x * k_sizex *sizez +y* sizez + z] = kernel[x][y][z];
			 }
		 }
	 }

	 double* dev_matrix1;//��������� �� ������ ���������� ������� 1
	 double* dev_kernel;//��������� �� ������ ����������  ����
	 double* dev_matrix2;//��������� �� ������ ����������  ������� 2
	 hipMalloc((void**)&dev_matrix1, sizeof(double) * m_sizex*m_sizey);//�������� ������ ��� ������� 1 �� ����������
	 hipMalloc((void**)&dev_kernel, sizeof(double) * k_sizex*k_sizey*sizez);//�������� ������ ��� ���� �� ����������
	 hipMalloc((void**)&dev_matrix2, sizeof(double) * m_sizex*m_sizey*sizez);//�������� ������ ��� ������� 2 �� ����������
	 hipMemcpy(dev_matrix1, matrixline, sizeof(double) * m_sizex*m_sizey, hipMemcpyHostToDevice);//�������� ������ � ������ ����������
	 hipMemcpy(dev_kernel, kernelline, sizeof(double) * k_sizex*k_sizey*sizez, hipMemcpyHostToDevice);//�������� ������ � ������ ����������

	
	 /* 28*28->32*32 �������=0 */
}


